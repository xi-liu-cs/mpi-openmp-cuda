#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RANGE 11.79

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(float * ad, float * bd, float * cd, int work_per_th, int n);

/**** end of the kernel declaration ***/

/* allocate ad, bd, and cd in the device */
void alloc(float ** ad_ptr, float ** bd_ptr, float ** cd_ptr, int n)
{
	hipMalloc(ad_ptr, n * sizeof(float));
	hipMalloc(bd_ptr, n * sizeof(float));
	hipMalloc(cd_ptr, n * sizeof(float));
}

/* send a, b, and c to the device */
void send(float * a, float * b, float * c, float * ad, float * bd, float * cd, int n)
{
	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bd, b, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cd, c, n * sizeof(float), hipMemcpyHostToDevice);
}

void print(float * a, int n)
{
	for(int i = 0; i < n; ++i)
		printf("%f ", a[i]);
	printf("\n");
}

int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	 

	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	start = clock();
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
		3. write the kernel, call it: vecGPU
		4. Call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to write the number of threads, blocks, etc and their geometry.
		5. Bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	alloc(&ad, &bd, &cd, n); /* 1 */
	send(a, b, c, ad, bd, cd, n); /* 2 */
	// int th_per_blk = 256, /* thread_per_block */
	// blk_per_grid = (n + th_per_blk - 1) / th_per_blk; /* block_per_grid = ceil(n / thread_per_block)*/
	// th_per_blk = 500;
	// blk_per_grid = 4; /* adjust numbers for report */
	// vecGPU<<<blk_per_grid, th_per_blk>>>(ad, bd, cd, n); /* 4 */

	int th_per_blk = 500, /* thread_per_block */
	blk_per_grid = 4, /* block_per_grid */
	work_per_th = n / (blk_per_grid * th_per_blk);
	// vecGPU<<<blk_per_grid, th_per_blk>>>(ad, bd, cd, work_per_th, n);
	vecGPU<<<1, 1>>>(ad, bd, cd, n, n);
	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost); /* 5 */
	hipFree(ad); hipFree(bd);	hipFree(cd); /* 6 */
	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	  if( fabsf(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		printf("Element %d in the result array does not match the sequential version\n", i);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/ /* 3 */
__global__ void vecGPU(float * ad, float * bd, float * cd, int work_per_th, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for(int work = 0; work < work_per_th; ++work)
	{
		if(i < n)
		{
			cd[i] += ad[i] * bd[i];
			++i;
		}
	}
}